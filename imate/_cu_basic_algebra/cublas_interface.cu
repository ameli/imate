/*
 *  SPDX-FileCopyrightText: Copyright 2021, Siavash Ameli <sameli@berkeley.edu>
 *  SPDX-License-Identifier: BSD-3-Clause
 *  SPDX-FileType: SOURCE
 *
 *  This program is free software: you can redistribute it and/or modify it
 *  under the terms of the license found in the LICENSE.txt file in the root
 *  directory of this source tree.
 */


// =======
// Headers
// =======

#include "./cublas_interface.h"


// ================
// cublas interface
// ================

/// \note      The implementation in the \c cu file is wrapped inside the
///            namepsace clause. This is not necessary in general, however, it
///            is needed to avoid the old gcc compiler error (this is a gcc
///            bug) which complains "no instance of function template matches
///            the argument list const float".

namespace cublas_interface
{

    // ===========
    // cublasXgemv (float)
    // ===========

    /// \brief A template wrapper for \c hipblasSgemv.
    ///

    template<>
    hipblasStatus_t cublasXgemv<float>(
            hipblasHandle_t handle,
            hipblasOperation_t trans,
            int m,
            int n,
            const float* alpha,
            const float* A,
            int lda,
            const float* x,
            int incx,
            const float* beta,
            float* y,
            int incy)
    {
        return hipblasSgemv(handle, trans, m, n, alpha, A, lda, x, incx, beta,
                           y, incy);
    }


    // ===========
    // cublasXgemv (double)
    // ===========

    /// \brief A template wrapper for \c hipblasDgemv.
    ///

    template<>
    hipblasStatus_t cublasXgemv<double>(
            hipblasHandle_t handle,
            hipblasOperation_t trans,
            int m,
            int n,
            const double* alpha,
            const double* A,
            int lda,
            const double* x,
            int incx,
            const double* beta,
            double* y,
            int incy)
    {
        return hipblasDgemv(handle, trans, m, n, alpha, A, lda, x, incx, beta,
                           y, incy);
    }



    // ===========
    // cublasXcopy (float)
    // ===========

    /// \brief  A template wrapper for \c hipblasScopy.
    ///

    template<>
    hipblasStatus_t cublasXcopy<float>(
            hipblasHandle_t handle,
            int n,
            const float* x,
            int incx,
            float* y,
            int incy)
    {
        return hipblasScopy(handle, n, x, incx, y, incy);
    }


    // ===========
    // cublasXcopy (double)
    // ===========

    /// \brief  A template wrapper for \c hipblasDcopy.
    ///

    template<>
    hipblasStatus_t cublasXcopy<double>(
            hipblasHandle_t handle,
            int n,
            const double* x,
            int incx,
            double* y,
            int incy)
    {
        return hipblasDcopy(handle, n, x, incx, y, incy);
    }


    // ===========
    // cublasXaxpy (float)
    // ===========

    /// \brief A template wrapper for \c hipblasSaxpy
    ///

    template<>
    hipblasStatus_t cublasXaxpy<float>(
            hipblasHandle_t handle,
            int n,
            const float *alpha,
            const float *x,
            int incx,
            float *y,
            int incy)
    {
        return hipblasSaxpy(handle, n, alpha, x, incx, y, incy);
    }


    // ===========
    // cublasXaxpy (double)
    // ===========

    /// \brief A template wrapper for \c hipblasDaxpy
    ///

    template<>
    hipblasStatus_t cublasXaxpy<double>(
            hipblasHandle_t handle,
            int n,
            const double *alpha,
            const double *x,
            int incx,
            double *y,
            int incy)
    {
        return hipblasDaxpy(handle, n, alpha, x, incx, y, incy);
    }


    // ==========
    // cublasXdot (float)
    // ==========

    /// \brief A template wrapper for \c hipblasSdot
    ///

    template<>
    hipblasStatus_t cublasXdot<float>(
            hipblasHandle_t handle,
            int n,
            const float *x,
            int incx,
            const float *y,
            int incy,
            float *result)
    {
        return hipblasSdot(handle, n, x, incx, y, incy, result);
    }


    // ==========
    // cublasXdot (double)
    // ==========

    /// \brief A template wrapper for \c hipblasDdot
    ///

    template<>
    hipblasStatus_t cublasXdot<double>(
            hipblasHandle_t handle,
            int n,
            const double *x,
            int incx,
            const double *y,
            int incy,
            double *result)
    {
        return hipblasDdot(handle, n, x, incx, y, incy, result);
    }


    // ===========
    // cublasXnrm2 (float)
    // ===========

    /// \brief A template wrapper to \c hipblasSnrm2
    ///

    template<>
    hipblasStatus_t cublasXnrm2<float>(
            hipblasHandle_t handle,
            int n,
            const float *x,
            int incx,
            float *result)
    {
        return hipblasSnrm2(handle, n, x, incx, result);
    }


    // ===========
    // cublasXnrm2 (double)
    // ===========

    /// \brief A template wrapper to \c hipblasDnrm2
    ///

    template<>
    hipblasStatus_t cublasXnrm2<double>(
            hipblasHandle_t handle,
            int n,
            const double *x,
            int incx,
            double *result)
    {
        return hipblasDnrm2(handle, n, x, incx, result);
    }


    // ===========
    // cublasXscal (float)
    // ===========

    /// \brief A template wrapper for \c hipblasSscal.
    ///

    template<>
    hipblasStatus_t cublasXscal<float>(
            hipblasHandle_t handle,
            int n,
            const float *alpha,
            float *x,
            int incx)
    {
        return hipblasSscal(handle, n, alpha, x, incx);
    }


    // ===========
    // cublasXscal (double)
    // ===========

    /// \brief A template wrapper for \c hipblasDscal.
    ///

    template<>
    hipblasStatus_t cublasXscal<double>(
            hipblasHandle_t handle,
            int n,
            const double *alpha,
            double *x,
            int incx)
    {
        return hipblasDscal(handle, n, alpha, x, incx);
    }
}  // namespace cublas_interface
