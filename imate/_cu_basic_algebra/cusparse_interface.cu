/*
 *  SPDX-FileCopyrightText: Copyright 2021, Siavash Ameli <sameli@berkeley.edu>
 *  SPDX-License-Identifier: BSD-3-Clause
 *  SPDX-FileType: SOURCE
 *
 *  This program is free software: you can redistribute it and/or modify it
 *  under the terms of the license found in the LICENSE.txt file in the root
 *  directory of this source tree.
 */


// =======
// Headers
// =======

#include "./cusparse_interface.h"
#include <cassert>  // assert

// ==================
// cusparse interface
// ==================

/// \note      The implementation in the \c cu file is wrapped inside the
///            namepsace clause. This is not necessary in general, however, it
///            is needed to avoid the old gcc compiler error (this is a gcc
///            bug) which complains "no instance of function template matches
///            the argument list const float".

namespace cusparse_interface
{

    // ======================
    // create cusparse matrix (float)
    // ======================

    /// \brief A template wrapper for \c hipsparseSpMatDescr_t for the \c float
    ///        precision data.

    template<>
    void create_cusparse_matrix<float>(
            hipsparseSpMatDescr_t& cusparse_matrix,
            const LongIndexType num_rows,
            const LongIndexType num_columns,
            const LongIndexType nnz,
            float* device_A_data,
            LongIndexType* device_A_indices,
            LongIndexType* device_A_index_pointer)
    {
        hipsparseStatus_t status = hipsparseCreateCsr(
                &cusparse_matrix, num_rows, num_columns, nnz,
                device_A_index_pointer, device_A_indices, device_A_data,
                HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

        assert(status == HIPSPARSE_STATUS_SUCCESS);
    }


    // ======================
    // create cusparse matrix (double)
    // ======================

    /// \brief A template wrapper for \c hipsparseSpMatDescr_t for the \c double
    ///        precision data.

    template<>
    void create_cusparse_matrix<double>(
            hipsparseSpMatDescr_t& cusparse_matrix,
            const LongIndexType num_rows,
            const LongIndexType num_columns,
            const LongIndexType nnz,
            double* device_A_data,
            LongIndexType* device_A_indices,
            LongIndexType* device_A_index_pointer)
    {
        hipsparseStatus_t status = hipsparseCreateCsr(
                &cusparse_matrix, num_rows, num_columns, nnz,
                device_A_index_pointer, device_A_indices, device_A_data,
                HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

        assert(status == HIPSPARSE_STATUS_SUCCESS);
    }


    // ======================
    // create cusparse vector (float)
    // ======================

    /// \brief   A template wrapper for \c hipsparseDnVecDescr_t for the
    ///          \c float precision data.
    ///
    /// \details Note that according to the cusparse documentation for the
    ///          function \c hipsparseCreateDnVec, it is safe to use
    ///          \c const_cast to cast the input vector.

    template<>
    void create_cusparse_vector<float>(
            hipsparseDnVecDescr_t& cusparse_vector,
            const LongIndexType vector_size,
            float* device_vector)
    {
        hipsparseStatus_t status = hipsparseCreateDnVec(
                &cusparse_vector, vector_size, device_vector, HIP_R_32F);

        assert(status == HIPSPARSE_STATUS_SUCCESS);
    }


    // ======================
    // create cusparse vector (double)
    // ======================

    /// \brief   A template wrapper for \c hipsparseDnVecDescr_t for the
    ///          \c double precision data.
    ///
    /// \details Note that according to the cusparse documentation for the
    ///          function \c hipsparseCreateDnVec, it is safe to use
    ///          \c const_cast to cast the input vector.

    template<>
    void create_cusparse_vector<double>(
            hipsparseDnVecDescr_t& cusparse_vector,
            const LongIndexType vector_size,
            double* device_vector)
    {
        hipsparseStatus_t status = hipsparseCreateDnVec(
                &cusparse_vector, vector_size, device_vector, HIP_R_64F);

        assert(status == HIPSPARSE_STATUS_SUCCESS);
    }


    // =======================
    // destroy cusparse matrix
    // =======================

    /// \brief Destroys cusparse matrix.
    ///

    void destroy_cusparse_matrix(
            hipsparseSpMatDescr_t& cusparse_matrix)
    {
        hipsparseStatus_t status = hipsparseDestroySpMat(cusparse_matrix);
        assert(status == HIPSPARSE_STATUS_SUCCESS);
    }


    // =======================
    // destroy cusparse vector
    // =======================

    /// \brief Destroys cusparse vector.
    ///

    void destroy_cusparse_vector(
            hipsparseDnVecDescr_t& cusparse_vector)
    {
        hipsparseStatus_t status = hipsparseDestroyDnVec(cusparse_vector);
        assert(status == HIPSPARSE_STATUS_SUCCESS);
    }


    // ===========================
    // cusparse matrix buffer size (float)
    // ===========================

    /// \brief A template wrapper for \c cusparseSpMat_buffersize for \c float
    ///        precision data. This function determines the buffer size needed
    ///        for matrix-vector multiplication using \c hipsparseSpMV. The
    ///        output is \c buffer_size variable.

    template<>
    void cusparse_matrix_buffer_size<float>(
            hipsparseHandle_t cusparse_handle,
            hipsparseOperation_t cusparse_operation,
            const float alpha,
            hipsparseSpMatDescr_t cusparse_matrix,
            hipsparseDnVecDescr_t cusparse_input_vector,
            const float beta,
            hipsparseDnVecDescr_t cusparse_output_vector,
            hipsparseSpMVAlg_t algorithm,
            size_t* buffer_size)
    {
        hipsparseStatus_t status = hipsparseSpMV_bufferSize(
                cusparse_handle, cusparse_operation, &alpha, cusparse_matrix,
                cusparse_input_vector, &beta, cusparse_output_vector,
                HIP_R_32F, algorithm, buffer_size);

        assert(status == HIPSPARSE_STATUS_SUCCESS);
    }


    // ===========================
    // cusparse matrix buffer size (double)
    // ===========================

    /// \brief A template wrapper for \c cusparseSpMat_buffersize for
    ///        \c double precision data. This function determines the buffer
    ///        size needed for matrix-vector multiplication using
    ///        \c hipsparseSpMV. The output is \c buffer_size variable.

    template<>
    void cusparse_matrix_buffer_size<double>(
            hipsparseHandle_t cusparse_handle,
            hipsparseOperation_t cusparse_operation,
            const double alpha,
            hipsparseSpMatDescr_t cusparse_matrix,
            hipsparseDnVecDescr_t cusparse_input_vector,
            const double beta,
            hipsparseDnVecDescr_t cusparse_output_vector,
            hipsparseSpMVAlg_t algorithm,
            size_t* buffer_size)
    {
        hipsparseStatus_t status = hipsparseSpMV_bufferSize(
                cusparse_handle, cusparse_operation, &alpha, cusparse_matrix,
                cusparse_input_vector, &beta, cusparse_output_vector,
                HIP_R_64F, algorithm, buffer_size);

        assert(status == HIPSPARSE_STATUS_SUCCESS);
    }


    // ===============
    // cusparse matvec (float)
    // ===============

    /// \brief A wrapper for \c hipsparseSpMV to perform sparse matrix-vector
    ///        multiplication uasing \c float precision data.

    template<>
    void cusparse_matvec<float>(
            hipsparseHandle_t cusparse_handle,
            hipsparseOperation_t cusparse_operation,
            const float alpha,
            hipsparseSpMatDescr_t cusparse_matrix,
            hipsparseDnVecDescr_t cusparse_input_vector,
            const float beta,
            hipsparseDnVecDescr_t cusparse_output_vector,
            hipsparseSpMVAlg_t algorithm,
            void* external_buffer)
    {
        hipsparseStatus_t status = hipsparseSpMV(cusparse_handle,
                                               cusparse_operation, &alpha,
                                               cusparse_matrix,
                                               cusparse_input_vector, &beta,
                                               cusparse_output_vector,
                                               HIP_R_32F, algorithm,
                                               external_buffer);

        assert(status == HIPSPARSE_STATUS_SUCCESS);
    }


    // ===============
    // cusparse matvec (double)
    // ===============

    /// \brief A wrapper for \c hipsparseSpMV to perform sparse matrix-vector
    ///        multiplication uasing \c double precision data.

    template<>
    void cusparse_matvec<double>(
            hipsparseHandle_t cusparse_handle,
            hipsparseOperation_t cusparse_operation,
            const double alpha,
            hipsparseSpMatDescr_t cusparse_matrix,
            hipsparseDnVecDescr_t cusparse_input_vector,
            const double beta,
            hipsparseDnVecDescr_t cusparse_output_vector,
            hipsparseSpMVAlg_t algorithm,
            void* external_buffer)
    {
        hipsparseStatus_t status = hipsparseSpMV(cusparse_handle,
                                               cusparse_operation, &alpha,
                                               cusparse_matrix,
                                               cusparse_input_vector, &beta,
                                               cusparse_output_vector,
                                               HIP_R_64F, algorithm,
                                               external_buffer);

        assert(status == HIPSPARSE_STATUS_SUCCESS);
    }
}  // namespace cusparse_interface
